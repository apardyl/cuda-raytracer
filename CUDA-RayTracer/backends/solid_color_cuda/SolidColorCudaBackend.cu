#include "hip/hip_runtime.h"
#include "SolidColorCudaBackend.h"
#if CUDA_ENABLED
#include "scene/Color.h"
#include <hip/hip_runtime.h>

const int BLOCK_SIZE = 32;

__global__ void renderSolidColor(Color *data, unsigned width, unsigned height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) {
        return;
    }

    data[(width * y + x)].red = 1;
    data[(width * y + x)].green = 0;
    data[(width * y + x)].blue = 0;
}

void SolidColorCudaBackend::doRender() {
    const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    const dim3 gridSize(
            width + (BLOCK_SIZE - 1) / BLOCK_SIZE,
            height + (BLOCK_SIZE - 1) / BLOCK_SIZE,
            1);
    renderSolidColor<<<gridSize, blockSize>>>(data, width, height);
}

#endif
