#include "hip/hip_runtime.h"
#include "RandomCudaBackend.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

const int BLOCK_SIZE = 32;
const int BYTES_PER_PIXEL = RandomCudaBackend::BYTES_PER_PIXEL;

__global__ void renderSolidColor(byte *data, unsigned width, unsigned height,
                                 unsigned seed, hiprandState_t* states) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) {
        return;
    }

    hiprandState_t *state = &states[y * width + x];
    hiprand_init(seed, (width * y + x), 0, state);

    data[(width * y + x) * BYTES_PER_PIXEL] = hiprand(state) % 255;
    data[(width * y + x) * BYTES_PER_PIXEL + 1] = hiprand(state) % 255;
    data[(width * y + x) * BYTES_PER_PIXEL + 2] = hiprand(state) % 255;
}

void RandomCudaBackend::doRender() {
    const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    const dim3 gridSize(
            width + (BLOCK_SIZE - 1) / BLOCK_SIZE,
            height + (BLOCK_SIZE - 1) / BLOCK_SIZE,
            1);

    hiprandState_t* states;
    hipMalloc((void**) &states, width * height * sizeof(hiprandState_t));

    renderSolidColor<<<gridSize, blockSize>>>(
            data, width, height, time(0), states);

    hipDeviceSynchronize();
    hipFree(states);
}
