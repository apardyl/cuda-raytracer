#include "CudaBackend.h"
#include <hip/hip_runtime.h>

CudaBackend::~CudaBackend() {
    hipFree(data);
}

Image CudaBackend::render() {
    doRender();
    hipDeviceSynchronize();
    return Image(width, height, data);
}

void CudaBackend::setResolution(unsigned width, unsigned height) {
    Backend::setResolution(width, height);
    hipFree(data);
    hipHostMalloc(&data, sizeof(byte) * width * height * BYTES_PER_PIXEL);
}
