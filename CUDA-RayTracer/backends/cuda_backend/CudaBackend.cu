#include "CudaBackend.h"
#if CUDA_ENABLED

#include <hip/hip_runtime.h>

CudaBackend::~CudaBackend() {
    hipFree(data);
}

Image CudaBackend::render() {
    doRender();
    hipDeviceSynchronize();
    return Image(width, height, data);
}

void CudaBackend::setResolution(unsigned width, unsigned height) {
    Backend::setResolution(width, height);
    hipFree(data);
    hipHostMalloc(&data, sizeof(Color) * width * height);
}

#endif //CUDA_ENABLED
